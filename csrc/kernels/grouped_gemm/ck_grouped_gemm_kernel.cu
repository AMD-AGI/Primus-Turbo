#include "hip/hip_runtime.h"
// Copyright (c) 2025, Advanced Micro Devices, Inc. All rights reserved.
//
// See LICENSE for license information.

#include "ck_grouped_gemm_kernel.h"

namespace primus_turbo {

template <typename ADataType, typename BDataType, typename CDataType, typename ALayout,
          typename BLayout, typename CLayout, typename TileConfig, typename AccDataType>
void CKGroupedGemmRunner<ADataType, BDataType, CDataType, ALayout, BLayout, CLayout, TileConfig,
                         AccDataType>::run(const ck_tile::stream_config &stream_cfg,
                                           const ck_tile::index_t group_num, void *args_ptr,
                                           const uint32_t num_cu) {

    constexpr int kBlockPerCu = 1;

    constexpr dim3 blocks = Kernel::BlockSize();
    dim3           grids  = Kernel::MaxOccupancyGridSize(stream_cfg);
    grids.x               = std::min(grids.x, num_cu);
    ck_tile::launch_kernel(
        stream_cfg, ck_tile::make_kernel<blocks.x, kBlockPerCu>(
                        Kernel{}, grids, blocks, 0,
                        ck_tile::cast_pointer_to_constant_address_space(args_ptr), group_num));
}

template <typename ADataType, typename BDataType, typename CDataType, typename AccDataType,
          typename ALayout, typename BLayout, typename CLayout>
std::unique_ptr<CKGroupedGemmRunnerInterFace>
get_ck_grouped_gemm_instance(const ck_tile::index_t group_num, const ck_tile::index_t m,
                             const ck_tile::index_t n, const ck_tile::index_t k) {
    using TileConfig = CKGroupedGemmTileCfg_256x256x64_32x32x16_2x2x1;
    using Runner = CKGroupedGemmRunner<ADataType, BDataType, CDataType, ALayout, BLayout, CLayout,
                                       TileConfig, AccDataType>;

    return std::make_unique<Runner>();
}

// ** FP16 **
// NT
template std::unique_ptr<CKGroupedGemmRunnerInterFace>
    get_ck_grouped_gemm_instance<ck_tile::half_t, ck_tile::half_t, ck_tile::half_t, float, RowMajor,
                                 ColMajor, RowMajor>(ck_tile::index_t, ck_tile::index_t,
                                                     ck_tile::index_t, ck_tile::index_t);
// NN
template std::unique_ptr<CKGroupedGemmRunnerInterFace>
    get_ck_grouped_gemm_instance<ck_tile::half_t, ck_tile::half_t, ck_tile::half_t, float, RowMajor,
                                 RowMajor, RowMajor>(ck_tile::index_t, ck_tile::index_t,
                                                     ck_tile::index_t, ck_tile::index_t);
// TN
template std::unique_ptr<CKGroupedGemmRunnerInterFace>
    get_ck_grouped_gemm_instance<ck_tile::half_t, ck_tile::half_t, ck_tile::half_t, float, ColMajor,
                                 RowMajor, RowMajor>(ck_tile::index_t, ck_tile::index_t,
                                                     ck_tile::index_t, ck_tile::index_t);

// ** BF16 **
// NT
template std::unique_ptr<CKGroupedGemmRunnerInterFace>
    get_ck_grouped_gemm_instance<ck_tile::bfloat16_t, ck_tile::bfloat16_t, ck_tile::bfloat16_t,
                                 float, RowMajor, ColMajor, RowMajor>(ck_tile::index_t,
                                                                      ck_tile::index_t,
                                                                      ck_tile::index_t,
                                                                      ck_tile::index_t);
// NN
template std::unique_ptr<CKGroupedGemmRunnerInterFace>
    get_ck_grouped_gemm_instance<ck_tile::bfloat16_t, ck_tile::bfloat16_t, ck_tile::bfloat16_t,
                                 float, RowMajor, RowMajor, RowMajor>(ck_tile::index_t,
                                                                      ck_tile::index_t,
                                                                      ck_tile::index_t,
                                                                      ck_tile::index_t);
// TN
template std::unique_ptr<CKGroupedGemmRunnerInterFace>
    get_ck_grouped_gemm_instance<ck_tile::bfloat16_t, ck_tile::bfloat16_t, ck_tile::bfloat16_t,
                                 float, ColMajor, RowMajor, RowMajor>(ck_tile::index_t,
                                                                      ck_tile::index_t,
                                                                      ck_tile::index_t,
                                                                      ck_tile::index_t);

} // namespace primus_turbo
