#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025 DeepSeek. All rights reserved.
 *
 * Modification Copyright© 2025 Advanced Micro Devices, Inc. All rights reserved.
 *
 * See LICENSE for license information.
 */

#include <cstring>

#include "configs.h"
#include "launch.h"
#include "primus_turbo/common.h"
#include "utils.h"

namespace primus_turbo::deep_ep {

namespace intranode {

template <int kNumRanks> __global__ void barrier(int **task_fifo_ptrs, int head, int rank) {
    barrier_device<kNumRanks>(task_fifo_ptrs, head, rank);
}

void barrier(int **task_fifo_ptrs, int head, int rank, int num_ranks, hipStream_t stream) {
#define BARRIER_LAUNCH_CASE(ranks)                                                                 \
    LAUNCH_KERNEL(&cfg, barrier<ranks>, task_fifo_ptrs, head, rank);                               \
    break

    SETUP_LAUNCH_CONFIG(1, kWarpSize, stream);
    SWITCH_RANKS(BARRIER_LAUNCH_CASE);
#undef BARRIER_LAUNCH_CASE
}

} // namespace intranode

namespace internode {

std::vector<uint8_t> get_unique_id() {
    PRIMUS_TURBO_CHECK(false, "not support");
    return {};
}

#
int init(const std::vector<uint8_t> &root_unique_id_val, int rank, int num_ranks,
         bool low_latency_mode) {
    PRIMUS_TURBO_CHECK(false, "not support");
    return 0;
}

void *alloc(size_t size, size_t alignment) {
    PRIMUS_TURBO_CHECK(false, "not support");
}

void free(void *ptr) {
    PRIMUS_TURBO_CHECK(false, "not support");
}

void barrier() {
    PRIMUS_TURBO_CHECK(false, "not support");
}

void finalize() {
    PRIMUS_TURBO_CHECK(false, "not support");
}
} // namespace internode

} // namespace primus_turbo::deep_ep
